#include "hip/hip_runtime.h"
#include <math.h>
#include "lcutil.h"
#include "timestamp.h"

// Kernel function with threads - one grid cell per thread
__global__ void kupdate(float* next, const float* prev, const int dim)
{
	// parameters for heat transfer
	struct Parms
	{ 
		float cx;
		float cy;
	} parms = {(float) 0.1, (float) 0.1};
	
	// Determine exact thread ID
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	// y -> row, x -> column, offset -> exact grid cell
	int y = index / (dim + 2);
	int x = index % (dim + 2);
	int offset = y * (dim + 2) + x;
	
	// Neighbours
	int left = offset - 1;
	int right = offset + 1;
	int top = offset - (dim + 2);
	int bottom = offset + (dim + 2);
	
	// make sure that thread's cell is not out of grid bounds & is inside the perimeter
	if (index < (dim + 2) * (dim + 2))
		if (y >= 1 && x >= 1 && y <= dim && x <= dim)
			next[offset] = prev[offset]
							+ parms.cx
								* (prev[left] + prev[right] - (float) 2.0 * prev[offset])
							+ parms.cy
								* (prev[top] + prev[bottom] - (float) 2.0 * prev[offset]);
}

extern "C" float update_gpu(float* next, const float* prev, const int dim, const int steps)
{
	int block_size;														// Number of threads per block
	int num_blocks;														// Number of blocks
	float* d_prev;														// Device source grid
	float* d_next;														// Device destination grid
	float* temp_grid;													// Two grids - one for before and one for after and temp points to "before"
	float* next_grid;													// next points to "after"
	int cur_grid;														// IF 0 -> source is current temperature grid ELSE dest ...
	timestamp s_time;
	float t_time;
	
	// Allocate memory for d_prev & d_next in the device
	CUDA_SAFE_CALL(hipMalloc((void**) &d_prev, sizeof(float) * ((dim + 2) * (dim + 2))));
	CUDA_SAFE_CALL(hipMalloc((void**) &d_next, sizeof(float) * ((dim + 2) * (dim + 2))));
	
	// Copy grids from CPU to device
	CUDA_SAFE_CALL(hipMemset(d_next, 0, sizeof(float) * ((dim + 2) * (dim + 2))));
	CUDA_SAFE_CALL(hipMemcpy(d_prev, prev, sizeof(float) * ((dim + 2) * (dim + 2)), hipMemcpyHostToDevice));
	
	// Create a block of threads (Threads per block - maximum 1024 threads per block for GTX 480)
	block_size = 512;
	dim3 threads(block_size);
	
	// Define number of blocks of block_size we need
	num_blocks = FRACTION_CEILING((float) (dim + 2) * (dim + 2), (float) block_size);
	dim3 blocks(num_blocks);
	
	cur_grid = 0;
	
	// define start time for update
	s_time = getTimestamp();
	
	for (int i = 0; i < steps; i++)
	{
		// set "before" and "after" grid
		temp_grid = (cur_grid == 0 ? d_prev : d_next);
		next_grid = (cur_grid != 0 ? d_prev : d_next);
		
		// update grid
		kupdate<<<blocks, threads>>>(next_grid, temp_grid, dim);
		
		// swap grids
		cur_grid = 1 - cur_grid;
	}
	
	// calculate elapsed time for update
	t_time = getElapsedtime(s_time);
	
	// determine errors & synchronize threads
	CUDA_SAFE_CALL(hipGetLastError());
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	
	// copy destination grid from device to cpu
	CUDA_SAFE_CALL(hipMemcpy(next, next_grid, sizeof(float) * ((dim + 2) * (dim + 2)), hipMemcpyDeviceToHost));
	
	// free device grids
	CUDA_SAFE_CALL(hipFree(d_prev));
	CUDA_SAFE_CALL(hipFree(d_next));
	
	return t_time;
}
